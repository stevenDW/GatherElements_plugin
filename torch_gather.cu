#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <cmath>

//#include "cuda_utils.h"
#include "torch_gather.h"
#include <hip/hip_runtime.h>
#include "cuda_util.h"
// namespace amirstan {
// namespace plugin {

using namespace amirstan::cuda;

using amirstan::cuda::TensorSize;
using amirstan::cuda::TensorStride;
// namespace cuda{
// #define CUDA_KERNEL_LOOP(i, n)                                 \
//   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
//        i += blockDim.x * gridDim.x)

// #define cudaCheckError()                                       \
//   {                                                            \
//     hipError_t e = hipGetLastError();                        \
//     if (e != hipSuccess) {                                    \
//       printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
//              hipGetErrorString(e));                           \
//       exit(0);                                                 \
//     }                                                          \
//   }

// const int CUDA_NUM_THREADS = 512;
// const int CUDA_WARP_SIZE = 32;
// const int CUDA_NUM_WARP = CUDA_NUM_THREADS / float(CUDA_WARP_SIZE);
// const int kMaxGridNum = 65535;

// inline int DIVUP(const int N, const int v) { return (N + v - 1) / v; }
// inline int GET_BLOCKS(const int N) {
//   return std::min(kMaxGridNum, DIVUP(N, CUDA_NUM_THREADS));
// }

// struct TensorSize {
//   int size[8];
//   int dims;
// };

// struct TensorStride {
//   size_t size[8];
//   int dims;
// };

// template <class value_type>
// void memcpyPermute(value_type *dst, const value_type *src, int *src_size,
//                    int *permute, int src_dim, hipStream_t stream = 0);

// template <typename T>
// void tensorMean(T *dst, T *src, int *src_size, bool *reduce_dims, int dims,
//                 hipStream_t stream = 0, void *workspace = nullptr);

// template <typename T>
// void tensorMeanVar(T *mean_dst, T *var_dst, const T *src, int *src_size,
//                    bool *reduce_dims, int dims, hipStream_t stream = 0,
//                    void *workspace = nullptr);

// template <typename T>
// void repeat_dims(T *dst, const T *src, const int *input_size,
//                  const int *repeatDims, int dims, hipStream_t stream = 0);

template <typename T, int nb_dims>
__global__ void torch_gather_kernel(T *__restrict__ dst,
                                    const T *__restrict__ src,
                                    const int *__restrict__ gather_table,
                                    int dim, TensorStride input_stride,
                                    TensorStride output_stride,
                                    int num_output) {
  size_t *__restrict__ src_stride = &(input_stride.size[0]);
  size_t *__restrict__ dst_stride = &(output_stride.size[0]);

  CUDA_KERNEL_LOOP(index, num_output) {
    int dst_index = index;
    int src_index = 0;
    const int gather_value = gather_table[dst_index];
#pragma unroll
    for (int i = 0; i < nb_dims; ++i) {
      const int dst_stride_i = dst_stride[i];
      const int src_stride_i = src_stride[i];
      int dim_index = dst_index / dst_stride_i;
      dst_index = dst_index % dst_stride_i;
      src_index += ((i != dim) ? dim_index : gather_value) * src_stride_i;
    }
    dst[index] = src[src_index];
  }
}
// }
template <typename T>
void torch_gather(T *output, const T *input, const int *index, int dim,
                  int *input_dims, int *index_dims, int nb_dims,
                  hipStream_t stream) {
  TensorSize ts_input_size;
  TensorStride input_stride;

  memcpy(&ts_input_size.size[0], input_dims, sizeof(int) * nb_dims);
  input_stride.size[nb_dims - 1] = 1;
  for (int i = nb_dims - 2; i >= 0; --i) {
    input_stride.size[i] = input_stride.size[i + 1] * ts_input_size.size[i + 1];
  }

  TensorSize ts_output_size;
  TensorStride output_stride;
  memcpy(&ts_output_size.size[0], index_dims, sizeof(int) * nb_dims);
  output_stride.size[nb_dims - 1] = 1;
  for (int i = nb_dims - 2; i >= 0; --i) {
    output_stride.size[i] =
        output_stride.size[i + 1] * ts_output_size.size[i + 1];
  }

  size_t num_output = output_stride.size[0] * ts_output_size.size[0];

  switch (nb_dims) {
    case 1:
      torch_gather_kernel<T, 1>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 2:
      torch_gather_kernel<T, 2>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 3:
      torch_gather_kernel<T, 3>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 4:
      torch_gather_kernel<T, 4>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 5:
      torch_gather_kernel<T, 5>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 6:
      torch_gather_kernel<T, 6>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 7:
      torch_gather_kernel<T, 7>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 8:
      torch_gather_kernel<T, 8>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 9:
      torch_gather_kernel<T, 9>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    case 10:
      torch_gather_kernel<T, 10>
          <<<GET_BLOCKS(num_output), CUDA_NUM_THREADS, 0, stream>>>(
              output, input, index, dim, input_stride, output_stride,
              num_output);
      break;
    default:
      break;
  }
}

template void torch_gather<float>(float *output, const float *input,
                                  const int *index, int dim, int *input_dims,
                                  int *index_dims, int nb_dims,
                                  hipStream_t stream);

template void torch_gather<half>(half *output, const half *input,
                                 const int *index, int dim, int *input_dims,
                                 int *index_dims, int nb_dims,
                                 hipStream_t stream);

template void torch_gather<int>(int *output, const int *input, const int *index,
                                int dim, int *input_dims, int *index_dims,
                                int nb_dims, hipStream_t stream);

//}  // namespace plugin
//}  // namespace amirstan
